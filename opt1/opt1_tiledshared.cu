#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
using namespace std;
#define TILE_WIDTH 16

//__constant__ float MASK_CONST[64*64];
__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int WblockRow = ceil((float)(W_out)/TILE_WIDTH);
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    extern __shared__ float shared_input[];
    float* shared_input_ptr = &shared_input[0];
    int shared_block_width = S*(TILE_WIDTH - 1) + K;

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define shared_3d(i2, i1, i0) shared_input[(i2) * (shared_block_width * shared_block_width) + (i1) * (shared_block_width) + i0]
    
    // Insert your GPU convolution kernel code here
    int w = TILE_WIDTH * (blockIdx.z % WblockRow) + threadIdx.x;
    int h = TILE_WIDTH * (blockIdx.z / WblockRow) + threadIdx.y;
    int w_s = TILE_WIDTH * (blockIdx.z % WblockRow);
    int h_s = TILE_WIDTH * (blockIdx.z / WblockRow);
    int b = blockIdx.x;
    int m = blockIdx.y;

    for(int c = 0; c < C; c++){
        for(int i = threadIdx.y; i < shared_block_width; i += TILE_WIDTH){
            for(int j = threadIdx.x; j < shared_block_width; j += TILE_WIDTH){
                if(((h_s * S + i) < H) && ((w_s * S + j) < W)){
                    shared_3d(c, i, j) = in_4d(b, c, (h_s * S) + i, (w_s * S) + j);
                }
                else{
                    shared_3d(c, i, j) = 0;
                }
            }
        }
    }
    __syncthreads();

    if(h < H_out && w < W_out){
        float value = 0;
        for(int c = 0; c < C; c++){
            for(int p = 0; p < K; p++){
                for(int q = 0; q < K; q++){
                    value += shared_3d(c, threadIdx.y * S + p, threadIdx.x * S + q) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = value;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
    #undef shared_3d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
   const int H_out = (H - K)/S + 1;
   const int W_out = (W - K)/S + 1;

   hipMalloc((void**) device_output_ptr, B * M * H_out * W_out * sizeof(float));
   hipMalloc((void**) device_input_ptr, B * C * H * W * sizeof(float));
   hipMalloc((void**) device_mask_ptr, M * C * K * K * sizeof(float));

   hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
   //hipMemcpyToSymbol(HIP_SYMBOL(MASK_CONST), host_mask, M * C * K * K * sizeof(float), 0, hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    int W_grid = ceil((float)((W - K)/S + 1)/TILE_WIDTH);
    int H_grid = ceil((float)((H - K)/S + 1)/TILE_WIDTH);
    int Y_grid = H_grid * W_grid;
    const int shared_block_width = (TILE_WIDTH - 1)*S + K;

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(B, M, Y_grid);
    int kernel_Size = C * shared_block_width * shared_block_width;

    conv_forward_kernel<<<dimGrid, dimBlock, kernel_Size * sizeof(float)>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, (B * M * ((H - K)/S + 1) * ((W - K)/S + 1))*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
