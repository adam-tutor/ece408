#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
using namespace std;
#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    __shared__ float tile_input[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tile_mask[TILE_WIDTH][TILE_WIDTH];

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    //int WblockRow = ceil((float)(W_out)/TILE_WIDTH);
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    //int w = TILE_WIDTH * (blockIdx.z % WblockRow) + threadIdx.x;
    //int h = TILE_WIDTH * (blockIdx.z / WblockRow) + threadIdx.y;
    int b = blockIdx.z;
    int m = blockIdx.y;
    float value = 0.0;

    int h_row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int w_col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int h_out = w_col / W_out;
    int w_out = w_col % W_out;
    for(int i = 0; i < ceil((1.0)*(C * K * K)/TILE_WIDTH); i++){
        if((h_row) < (M) && (i * TILE_WIDTH + threadIdx.x) < (C*K*K)){
            tile_mask[threadIdx.y][threadIdx.x] = mask[h_row * C * K * K + i * TILE_WIDTH + threadIdx.x];
        }
        else{
            tile_mask[threadIdx.y][threadIdx.x] = 0;
        }
        if((i * TILE_WIDTH + threadIdx.y < (C * K * K)) && ((w_col) < (H_out * W_out))){
            int c = (i * TILE_WIDTH + threadIdx.y)/(K*K);
            
            int p = (i * TILE_WIDTH + threadIdx.y - (C * K * K))/K;
            int q = (i * TILE_WIDTH + threadIdx.y - (C * K * K))%K;

            if(((h_out * S + p) < H) && ((w_out*S+q) < W)){
                tile_input[threadIdx.y][threadIdx.x] = in_4d(b, c, h_out * S + p, w_out * S + q);
            }
            else{
                tile_input[threadIdx.y][threadIdx.x] = 0;
            }
        }
        else{
            tile_input[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        if((h_row < M) && (w_col < (H_out * W_out))){
            for(int k = 0; k < TILE_WIDTH; k++){
                value += tile_mask[threadIdx.y][k] * tile_input[k][threadIdx.x];
            }
        }
        __syncthreads();
    }
    if((h_row < M) && (w_col < (H_out * W_out))){
        //output[blockIdx.z * (M * (H_out * W_out)) + h_row * (H_out * W_out) + w_col] = value;
        out_4d(blockIdx.z, h_row, w_col / W_out, w_col % W_out) = value;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
   const int H_out = (H - K)/S + 1;
   const int W_out = (W - K)/S + 1;

   hipMalloc((void**) device_output_ptr, B * M * H_out * W_out * sizeof(float));
   hipMalloc((void**) device_input_ptr, B * C * H * W * sizeof(float));
   hipMalloc((void**) device_mask_ptr, M * C * K * K * sizeof(float));

   hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_grid = ceil((1.0 * H_out * W_out)/TILE_WIDTH);
    int H_grid = ceil((1.0 * M)/TILE_WIDTH);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(W_grid, H_grid, B);

    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize();
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, (B * M * ((H - K)/S + 1) * ((W - K)/S + 1))*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
