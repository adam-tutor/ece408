#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include "hip/hip_fp16.h"
using namespace std;
#define TILE_WIDTH 20

__global__ void conv_forward_kernel(half *output, const half *input, const half *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int WblockRow = ceil((float)(W_out)/TILE_WIDTH);
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int w = TILE_WIDTH * (blockIdx.z % WblockRow) + threadIdx.x;
    int h = TILE_WIDTH * (blockIdx.z / WblockRow) + threadIdx.y;
    int b = blockIdx.x;
    int m = blockIdx.y;
    int h_stride = h * S;
    int w_stride = w * S;

    if(h < H_out && w < W_out){
        half value = 0.0;
        for(int c = 0; c < C; c++){
            for(int p = 0; p < K; p++){
                for(int q = 0; q < K; q++){
                   value += in_4d(b, c, h * S + p, w * S + q) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = value;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__global__ void FloatToHalf(const float *input, half *output, const int size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = index; i < size; i += blockDim.x * gridDim.x){
        output[i] = __float2half(input[i]);
    }
}

__global__ void HalfToFloat(half *input, float *output, const int size){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = index; i < size; i += blockDim.x * gridDim.x){
        output[i] = __half2float(input[i]);
    }
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
   const int H_out = (H - K)/S + 1;
   const int W_out = (W - K)/S + 1;

   int input_size = B * C * H * W;
    int mask_size = M * C * K * K;
    int output_size = B * M * H_out * W_out;

   hipMalloc((void**) device_output_ptr, output_size * sizeof(float));
   hipMalloc((void**) device_input_ptr, input_size * sizeof(float));
   hipMalloc((void**) device_mask_ptr, mask_size * sizeof(float));

   hipMemcpy(*device_input_ptr, host_input, input_size * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    int W_grid = ceil(1.0 * W/TILE_WIDTH);
    int H_grid = ceil(1.0 * H/TILE_WIDTH);
    int Y_grid = H_grid * W_grid;

    half* half_device_input;
    half* half_device_mask;
    half* half_device_output;

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    int input_size = B * C * H * W;
    int mask_size = M * C * K * K;
    int output_size = B * M * H_out * W_out;

    hipMalloc(&half_device_input, input_size * sizeof(half));
    hipMalloc(&half_device_mask, mask_size * sizeof(half));
    hipMalloc(&half_device_output, output_size * sizeof(half));

    dim3 dimHalfBlock(1024, 1, 1);
    dim3 dimHalfGrid(TILE_WIDTH, 1, 1);

    FloatToHalf<<<dimHalfGrid, dimHalfBlock>>>(device_input, half_device_input, input_size);
    hipDeviceSynchronize();
    FloatToHalf<<<dimHalfGrid, dimHalfBlock>>>(device_mask, half_device_mask, mask_size);
    hipDeviceSynchronize();

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(B, M, Y_grid);

    conv_forward_kernel<<<dimGrid, dimBlock>>>(half_device_output, half_device_input, half_device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize();

    dim3 dimFloatBlock(1024, 1, 1);
    dim3 dimFloatGrid(TILE_WIDTH, 1, 1);
    HalfToFloat<<<dimFloatGrid, dimFloatBlock>>>(half_device_output, device_output, output_size);
    hipDeviceSynchronize();

    hipFree(half_device_input);
    hipFree(half_device_mask);
    hipFree(half_device_output);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, (B * M * ((H - K)/S + 1) * ((W - K)/S + 1))*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
